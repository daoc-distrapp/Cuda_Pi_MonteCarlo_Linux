#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

static const long BLOCKS = 256;
static const long THREAD_X_BLOCK = 256;
static const long ITER_X_THREAD = 1024 * 64;

__global__
void piMC(long long *blockCounter) { // blockCounter debe tener un contador por cada bloque

	// Debe haber un contador por cada hilo en el bloque (compartido  en el bloque)
	__shared__ long long threadCounter[THREAD_X_BLOCK];

	// ID de la thread
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	// Inicializa el RNG
	hiprandState_t rng;
	hiprand_init(clock64(), id, 0, &rng);

	// Inicializa el contador
	threadCounter[threadIdx.x] = 0;

	// Calcula los puntos dentro del círculo
	for (int i = 0; i < ITER_X_THREAD; i++) {
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		threadCounter[threadIdx.x] += 1 - int(x * x + y * y);
	}

	// La primera thread en cada bloque suma los contadores individuales en el de bloque
	if (threadIdx.x == 0) {
		// Inicializa el contador de este bloque
		blockCounter[blockIdx.x] = 0;
		// Suma los contadores de thread en el de bloque
		for (int i = 0; i < THREAD_X_BLOCK; i++) {
			blockCounter[blockIdx.x] += threadCounter[i];
		}
	}
}

int main(void) {
	// Crea el buffer para los contadores de bloque en el host
	long long *blockCounter = (long long*)malloc(sizeof(long long) * BLOCKS);

	// Crea el buffer para los contadores de bloque en la GPU
	long long *gpuBlockCounter;
	hipMalloc(&gpuBlockCounter, sizeof(long long) * BLOCKS);

	// Ejecuta la kernel
	piMC<<<BLOCKS, THREAD_X_BLOCK>>>(gpuBlockCounter);

	// Recupera el resultado desde la GPU y lo pone en el buffer del host
	hipMemcpy(blockCounter, gpuBlockCounter, sizeof(long long) * BLOCKS, hipMemcpyDeviceToHost);

	// Suma los contadores y calcula PI
	long long total = 0;
	for (int i = 0; i < BLOCKS; i++) {
		total += blockCounter[i];
	}
	long long iters = BLOCKS * THREAD_X_BLOCK * ITER_X_THREAD;
	printf("Aproximado con %lld iteraciones\n", iters);
	printf("%lld puntos dentro del círculo\n", total);
	printf("PI= %f\n", 4.0 * (double)total/(double)iters);

	// Libera los recursos
	hipFree(gpuBlockCounter);
	free(blockCounter);

	return 0;
}
